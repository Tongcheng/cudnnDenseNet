#include "hipDNN.h"
#include <stdio.h>
#include <stdlib.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>

float** GPU_filterDeploy(float** filter_host,int numTransform,int initChannel,int growthRate,int N,int filter_H,int filter_W){
    float** output_ptrs = new float*[numTransform];
    for (int transformIdx=0;transformIdx < numTransform;++transformIdx){
        float* output_gpu_local;
	int outputNumChannels = growthRate;
	int inputNumChannels = transformIdx * growthRate + initChannel;
	int localSize_bytes = outputNumChannels * inputNumChannels * H * W * sizeof(float);
        hipMalloc(&output_gpu_local,localSize_bytes);
	hipMemcpy(output_gpu_local,filter_host[transformIdx],localSize_bytes,hipMemcpyHostToDevice);
        output_ptrs[transformIdx] = output_gpu_local; 
    }
    return output_ptrs;
}

float** GPU_miscDeploy(float* BNScaler_host,float* BNBias_host,int numTransform,int initChannel,int growthRate,int N,int H,int W,int workspaceSize){
    float** output_ptrs = new float*[10];
    int totalNumChannel = initChannel+numTransform*growthRate;  
    //index 0 is BN_Scaler_Vec
    float* output_BN_Scaler;
    hipMalloc(&output_BN_Scaler,totalNumChannel*sizeof(float));
    hipMemcpy(output_BN_Scaler,BNScaler_host,totalNumChannel*sizeof(float),hipMemcpyHostToDevice);
    output_ptrs[0] = output_BN_Scaler;
    //index 1 is BN_Bias_Vec
    float* output_BN_Bias;
    hipMalloc(&output_BN_Bias,totalNumChannel*sizeof(float));
    hipMemcpy(output_BN_Bias,);
    output_ptrs[1] = output_BN_Bias;
    //index 2 is ResultRunningMean
    float* output_ResultRunningMean;
    hipMalloc(&output_ResultRunningMean,totalNumChannel*sizeof(float));
    hipMemset(output_ResultRunningMean,0,totalNumChannel*sizeof(float));
    output_ptrs[2] = output_ResultRunningMean;    
    //index 3 is ResultRunningVariance
    float* output_ResultRunningVariance;
    hipMalloc(&output_ResultRunningVariance,totalNumChannel*sizeof(float));
    hipMemset(output_ResultRunningVariance,0,totalNumChannel*sizeof(float)); 
    output_ptrs[3] = output_ResultRunningVariance; 
    //index 4 is ResultSaveMean
    float* output_ResultSaveMean;
    hipMalloc(&output_ResultSaveMean,totalNumChannel*sizeof(float));
    hipMemset(output_ResultSaveMean,0,totalNumChannel*sizeof(float));
    output_ptrs[4] = output_ResultSaveMean;
    //index 5 is ResultSaveInvVariance
    float* output_ResultSaveInvVariance;
    hipMalloc(&output_ResultSaveInvVariance,totalNumChannel*sizeof(float));
    hipMemset(output_ResultSaveInvVariance,0,totalNumChannel*sizeof(float));
    output_ptrs[5] = output_ResultSaveInvVariance;
    //index 6 is postConv_dataRegion
    int postSize = N*(initChannel+growthRate*numTransform)*H*W*sizeof(float); 
    float* postConv_dataPtr;
    hipMalloc(&postConv_dataPtr,postSize); 
    hipMemset(postConv_dataPtr,0,postSize);
    output_ptrs[6] = postConv_dataPtr; 
    //index 7 is postBN_dataRegion
    float* postBN_dataPtr;
    hipMalloc(&postBN_dataPtr,postSize);
    hipMemset(postBN_dataPtr,0,postSize);
    output_ptrs[7] = postBN_dataPtr;
    //index 8 is postReLU_dataRegion
    float* postReLU_dataPtr;
    hipMalloc(&postReLU_dataPtr,postSize);
    hipMemset(postReLU_dataPtr,0,postSize);
    output_ptrs[8] = postReLU_dataPtr; 
    //index 9 is workspace
    float* workspacePtr;
    hipMalloc(&workspacePtr,workspaceSize);
    hipMemset(workspacePtr,0,workspaceSize);
    output_ptrs[9] = workspacePtr; 
    //done and return
    return output_ptrs;  
}

/*DenseLayer: For each small transition within DenseLayer, do BN->ReLU->Convolution*/
//Input: # of channel = k0 + k(Order - 1)
//Output: # of channel = k
//testMode: 1 if test, 0 if train
//trainCycleIdx: the idx for current training cycle, related to EMA of BN, doesn't matter if in test
//BNScalerVec, BNBiasVec: Scaler and Bias per channel.
//resultRunningMean, resultRunningVariance: per channel.
//resultSaveMean, resultSaveInvVariance: null in testing phase.
//numTransition: number of BN->ReLU->Convolutions
//filter_transform: filter_transform on cpu,filter_transform[i] is on gpu 
void DenseBlockForward(int initChannel,int growthRate,int numTransition,
  int N,int H,int W,int pad_h,int pad_w,int conv_verticalStride,int conv_horizentalStride,
  int testMode, int trainCycleIdx,
  float* BNScalerVec, float* BNBiasVec,float* resultRunningMean, float* resultRunningVariance, float* resultSaveMean, float* resultSaveInvVariance, 
  float* postConv_dataRegion, float* postBN_dataRegion, float* postReLU_dataRegion,
  float** filter_transform,int filter_H,int filter_W,
  float* workspace_gpu,int workspaceSize
    ){
    hipdnnHandle_t* handlePtr = new hipdnnHandle_t;
    hipdnnCreate(handlePtr);
    float* oneScalerPtr = new float[1]; oneScalerPtr[0] = 1.0;
    float* zeroScalerPtr = new float[1]; zeroScalerPtr[0] = 0.0; 
    for (int transitionIdx=0;transitionIdx < numTransition;++transitionIdx){
	//BN transform
	hipdnnTensorDescriptor_t* BN_x_Descriptor = new hipdnnTensorDescriptor_t;
   	hipdnnTensorDescriptor_t* BN_y_Descriptor = new hipdnnTensorDescriptor_t;
    	hipdnnTensorDescriptor_t* BN_param_Descriptor = new hipdnnTensorDescriptor_t;
        hipdnnTensorDescriptor_t* ReLU_y_Descriptor = new hipdnnTensorDescriptor_t;
        hipdnnCreateTensorDescriptor(BN_x_Descriptor);
    	hipdnnCreateTensorDescriptor(BN_y_Descriptor);
        hipdnnCreateTensorDescriptor(BN_param_Descriptor);
        hipdnnCreateTensorDescriptor(ReLU_y_Descriptor);
	//same channel size pre and post Mapping
        int numChannelTransform = growthRate;
        if (transitionIdx==0){numChannelTransform = initChannel;}
	
        hipdnnSetTensor4dDescriptorEx(*BN_x_Descriptor,HIPDNN_DATA_FLOAT,N,numChannelTransform,H,W,
          (numTransition*growthRate+initChannel)*H*W,H*W,W,1
        );
        hipdnnSetTensor4dDescriptorEx(*BN_y_Descriptor,HIPDNN_DATA_FLOAT,N,numChannelTransform,H,W,
	  (numTransition*growthRate+initChannel)*H*W,H*W,W,1
	);
        hipdnnSetTensor4dDescriptorEx(*ReLU_y_Descriptor,HIPDNN_DATA_FLOAT,N,numChannelTransform,H,W,
	  (numTransition*growthRate+initChannel)*H*W,H*W,W,1
	);      
 
	if (transitionIdx==0){
            hipdnnSetTensor4dDescriptor(*BN_param_Descriptor,HIPDNN_TENSOR_NCHW,HIPDNN_DATA_FLOAT,1,initChannel,1,1);
        } else {
	    hipdnnSetTensor4dDescriptor(*BN_param_Descriptor,HIPDNN_TENSOR_NCHW,HIPDNN_DATA_FLOAT,1,growthRate,1,1);
        }
        int channelsBefore_noself = (transformIdx==0?0:initChannel+(transformIdx-1) * growth);
	int channelsBefore_self = initChannel + transitionIdx * growth;
        float* BN_x_ptr = postConv_dataRegion+channelsBefore_noself*H*W;
	float* BN_y_ptr = postBN_dataRegion+channelsBefore_noself*H*W;
        float* BN_scaler_local = BNScalerVec + channelsBefore_noself;
	float* BN_bias_local = BNBiasVec + channelsBefore_noself; 
        float* BN_mean_local = resultRunningMean + channelsBefore_noself; 
	float* BN_var_local = resultRunningVariance + channelsBefore_noself;
	if (testMode){
	    hipdnnBatchNormalizationForwardInference(*handlePtr,HIPDNN_BATCHNORM_SPATIAL,oneScalerPtr,zeroScalerPtr,*BN_x_Descriptor,BN_x_ptr,*BN_y_Descriptor,BN_y_ptr,*BN_param_Descriptor,BN_scaler_local,BN_bias_local,BN_mean_local,BN_var_local,HIPDNN_BN_MIN_EPSILON);
        }
	else {
	    float* resultSaveMean_local = resultSaveMean + channelsBefore_noself;
	    float* resultSaveInvVariance_local = resultSaveInvVariance + channelsBefore_noself;
            float exponentialMovingAverageFactor = 1.0/(1+trainCycleIdx);
	    hipdnnBatchNormalizationForwardTraining(*handlePtr,HIPDNN_BATCHNORM_SPATIAL,oneScalerPtr,zeroScalerPtr,*BN_x_Descriptor,BN_x_ptr,*BN_y_Descriptor,BN_y_ptr,*BN_param_Descriptor,BN_scaler_local,BN_bias_local,BN_mean_local,BN_var_local,HIPDNN_BN_MIN_EPSILON,resultSaveMean_local,resultSaveInvVariance_local);
        }
	//ReLU transform
        float* ReLU_y_ptr = postBN_dataRegion+channelsBefore_noself*H*W; 
	hipdnnActivationDescriptor_t* activationDescPtr = new hipdnnActivationDescriptor_t;
	hipdnnCreateActivationDescriptor(activationDescPtr);
	hipdnnSetActivationDescriptor(*activationDescPtr,HIPDNN_ACTIVATION_RELU,HIPDNN_NOT_PROPAGATE_NAN,0.0);
        hipdnnActivationForward(*handlePtr,*activationDescPtr,oneScalerPtr,*BN_y_Descriptor,BN_y_ptr,zeroScalerPtr,ReLU_y_Descriptor,ReLU_y_ptr);
        //Convolution
	//Convolution::tensor Descriptor
        hipdnnTensorDescriptor_t* Conv_x_Descriptor = new hipdnnTensorDescriptor_t;
        hipdnnTensorDescriptor_t* Conv_y_Descriptor = new hipdnnTensorDescriptor_t; 
        hipdnnCreateTensorDescriptor(Conv_x_Descriptor);
        hipdnnCreateTensorDescriptor(Conv_y_Descriptor);
	hipdnnSetTensor4dDescriptorEx(*Conv_x_Descriptor,HIPDNN_DATA_FLOAT,N,channelsBefore_self,H,W,(numTransition*growthRate+initChannel)*H*W,H*W,W,1);
        hipdnnSetTensor4dDescriptorEx(*Conv_y_Descriptor,HIPDNN_DATA_FLOAT,N,growthRate,H,W,(numTransition*growthRate+initChannel)*H*W,H*W,W,1);
	//Convolution::tensor Ptr
        int delayChannels = initChannel+growthRate*transitionIdx;
	float* conv_x_local = postConv_dataRegion; 
	float* conv_y_local = postConv_dataRegion + delayChannels*H*W;
        //Convolution::filter Descriptor
	hipdnnFilterDescriptor_t* filterDescriptor = new hipdnnFilterDescriptor_t;
	hipdnnCreateFilterDescriptor(filterDescriptor);
	hipdnnSetFilter4dDescriptor(*filterDescriptor,HIPDNN_DATA_FLOAT,HIPDNN_TENSOR_NCHW,growthRate,channelsBefore_self,filter_H,filter_W);   	
	//Convolution::convolution Descriptor	
        hipdnnConvolutionDescriptor_t* convolutionDescriptor = new hipdnnConvolutionDescriptor_t;
        hipdnnCreateConvolutionDescriptor(convolutionDescriptor);
        hipdnnSetConvolution2dDescriptor(*convolutionDescriptor,pad_h,pad_w,conv_verticalStride,conv_horizentalStride,1,1,HIPDNN_CONVOLUTION);  
       
        hipdnnConvolutionForward(*handlePtr,oneScalerPtr,Conv_x_Descriptor,conv_x_local,filterDescriptor,filter_transform[transformIdx],convolutionDescriptor,HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,workspace_gpu,workspace,zeroScalerPtr,Conv_y_Descriptor,conv_y_local); 
    }
 
}

void DenseBlockBackward(float* postConv_data,float* postBN_data,float* postReLU_data,
  float* postConv_grad,float* postBN_grad,float* postReLU_grad,
  float* BNscaler_grad, float* BNbias_grad, float** filter_grad,
  float* BNscaler_data, float* BNbias_data, float** filter_data,
  float* resultSaveMean, float* resultSaveInvVariance,
  int numTransition,int N,int H,int W,int initChannel,int growthRate,
  int pad_h,int pad_w,int conv_verticalStride,int conv_horizentalStride,
  int filter_H,int filter_W,
  float* workspace_gpu, int workspaceSize 
){
    hipdnnHandle_t* handlePtr = new hipdnnHandle_t;
    hipdnnCreate(handlePtr);
    float* oneScalePtr = new float[1]; oneScalePtr[0] = 1.0;
    float* zeroScalePtr = new float[1]; zeroScalePtr[0] = 0.0;
    
    for (int transitionIdx = numTransition-1;transitionIdx>=0;--transitionIdx){
        int channelsBefore_self = initChannels + transitionIdx * growthRate; 
        int channelsBefore_noself = (transitionIdx>0?initChannel:0)+(transitionIdx-1)*growthRate;
	//Conv backward::Preparation
        hipdnnFilterDescriptor_t* filterDesc = new hipdnnFilterDescriptor_t;
        hipdnnCreateFilterDescriptor(filterDesc);
	hipdnnSetFilter4dDescriptor(*filterDesc,HIPDNN_DATA_FLOAT,HIPDNN_TENSOR_NCHW,growthRate,channelsBefore_self,filter_H,filter_W);
        hipdnnConvolutionDescriptor_t* convolutionDescriptor = new hipdnnConvolutionDescriptor_t;
        hipdnnCreateConvolutionDescriptor(convolutionDescriptor);
        hipdnnSetConvolution2dDescriptor(*convolutionDescriptor,pad_h,pad_w,conv_verticalStride,conv_horizentalStride,1,1,HIPDNN_CONVOLUTION);  
        hipdnnTensorDescriptor_t* Conv_x_Descriptor = new hipdnnTensorDescriptor_t;
        hipdnnCreateTensorDescriptor(Conv_x_Descriptor);
        hipdnnSetTensor4dDescriptorEx(*Conv_x_Descriptor,HIPDNN_DATA_FLOAT,N,channelsBefore_self,H,W,(numTransition*growthRate+initChannel)*H*W,H*W,W,1);
        hipdnnTensorDescriptor_t* Conv_y_Descriptor = new hipdnnTensorDescriptor_t;
        hipdnnCreateTensorDescriptor(Conv_y_Descriptor);
        hipdnnSetTensor4dDescriptorEx(*Conv_y_Descriptor,HIPDNN_DATA_FLOAT,N,growthRate,H,W,(numTransition*growthRate+initChannel)*H*W,H*W,W,1);
        //Conv backward::filter grad
        float* filterGrad_local = filter_grad[transitionIdx];
        float* conv_x_ptr = postReLU_data;
        float* conv_dy_ptr = postConv_grad + channelsBefore_self*H*W;
        hipdnnConvolutionBackwardFilter(*handlePtr,oneScalePtr,*Conv_x_Descriptor,conv_x_ptr,
          *Conv_y_Descriptor,conv_dy_ptr,*convolutionDescriptor,HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_1,workspace_gpu,workspaceSize,
	  zeroScalePtr,*filterDesc,filterGrad_local
        );       
	//Conv backward::data grad
        float* filterData_local = filter_data[transitionIdx]; 
	hipdnnConvolutionBackwardData(*handlePtr,oneScalePtr,*filterDesc,filterData_local,
            *Conv_y_Descriptor,conv_dy_ptr,*convolutionDescriptor,HIPDNN_CONVOLUTION_BWD_DATA_ALGO_1,workspace_gpu,workspaceSize,oneScalePtr,*Conv_x_Descriptor,postReLU_grad
        ); 	
	//ReLU backward
        int numChannelTransform = (transformIdx==0?initChannel:growthRate);
        hipdnnActivationDescriptor_t* activationDescPtr = new hipdnnActivationDescriptor_t;
	hipdnnCreateActivationDescriptor(activationDescPtr);
	hipdnnSetActivationDescriptor(*activationDescPtr,HIPDNN_ACTIVATION_RELU,HIPDNN_NOT_PROPAGATE_NAN,0.0);
        hipdnnTensorDescriptor_t* Bijective_Tensor_Descriptor = new hipdnnTensorDescriptor_t;
        hipdnnCreateTensorDescriptor(Bijective_Tensor_Descriptor);
    	hipdnnSetTensor4dDescriptorEx(*Bijective_Tensor_Descriptor,HIPDNN_DATA_FLOAT,N,numChannelTransform,H,W,
	  (numTransition*growthRate+initChannel)*H*W,H*W,W,1
	);

	float* ReLU_y_ptr = postReLU_data + channelsBefore_noself*H*W;
	float* ReLU_x_ptr = postBN_data + channelsBefore_noself*H*W; 
	float* ReLU_dy_ptr = postReLU_grad + channelsBefore_noself*H*W;
	float* ReLU_dx_ptr = postBn_grad + channelsBefore_noself*H*W; 
	hipdnnActivationBackward(*handlePtr,*activationDescPtr,oneScalePtr,*Bijective_Tensor_Descriptor,ReLU_y_ptr,*Bijective_Tensor_Descriptor,ReLU_dy_ptr,*Bijective_Tensor_Descriptor,ReLU_x_ptr,zeroScalePtr,*Bijective_Tensor_Descriptor,ReLU_dx_ptr);	
	//BN backward
	float* BN_x_ptr = postConv_data + channelsBefore_noself*H*W;
	float* BN_dx_ptr = postConv_grad + channelsBefore_noself*H*W;
        hipdnnTensorDescriptor_t* BN_param_Descriptor = new hipdnnTensorDescriptor_t;
        hipdnnCreateTensorDescriptor(BN_param_Descriptor);
    	hipdnnSetTensor4dDescriptor(*BN_param_Descriptor,HIPDNN_TENSOR_NCHW,HIPDNN_DATA_FLOAT,1,numChannelTransform,1,1);
        float* BNscaler_data_localPtr = BNscaler_data + channelsBefore_noself;
	float* BNscaler_grad_localPtr = BNscaler_grad + channelsBefore_noself;
	float* Bnbias_grad_localPtr = BNbias_grad + channelsBefore_noself;      
        float* saveMean_local = resultSaveMean + channelsBefore_noself;
	float* saveInvVar_local = resultSaveInvVariance + channelsBefore_noself; 
        hipdnnBatchNormalizationBackward(*handlePtr,HIPDNN_BATCHNORM_SPATIAL,oneScalePtr,zeroScalePtr,oneScalePtr,zeroScalePtr,*BijectiveTensorDescriptor,BN_x_ptr,*BijectiveTensorDescriptor,ReLU_dx_ptr,*BijectiveTensorDescriptor,BN_dx_ptr,*BN_param_Descriptor,BNscaler_data_localPtr,BNscaler_grad_localPtr,BNbias_grad_localPtr,HIPDNN_BN_MIN_EPSILON,saveMean_local,saveInvVar_local); 
    }
}




